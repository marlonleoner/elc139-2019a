
#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdlib>
#include <stdio.h>
#include <math.h>
#include <assert.h>
#include <sys/time.h>

__global__ void fPixelGenerator(int width, int frames, unsigned char* pic) {

   int blockID = blockIdx.x;
   int tBlocks = gridDim.x;

   int fromBlocks = (blockID)     * (frames / tBlocks);
   int toBlocks   = (blockID + 1) * (frames / tBlocks);

   int threadID = threadIdx.x;
   int tThreads = blockDim.x;

   int fromThreads = (threadID)     * (width / tThreads);
   int toThreads   = (threadID + 1) * (width / tThreads);

   for (int frame = fromBlocks; frame < toBlocks; frame++) {
      for (int row = fromThreads; row < toThreads; row++) {
         for (int col = 0; col < width; col++) {
            float fx = col - 1024/2;
            float fy = row - 1024/2;
            float d = sqrtf( fx * fx + fy * fy );
            unsigned char color = (unsigned char) (160.0f + 127.0f * cos(d/10.0f - frame/7.0f) / (d/50.0f + 1.0f));

            pic[frame * width * width + row * width + col] = (unsigned char) color;
         }
      }
   }
}

static void writeBMP(const int x, const int y, const unsigned char* const bmp, const char* const name) {

   const unsigned char bmphdr[54] = {66, 77, 255, 255, 255, 255, 0, 0, 0, 0, 54, 4, 0, 0, 40, 0, 0, 0, 255, 255, 255, 255, 255, 255, 255, 255, 1, 0, 8, 0, 0, 0, 0, 0, 255, 255, 255, 255, 196, 14, 0, 0, 196, 14, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0};
   unsigned char hdr[1078];
   int i, j, c, xcorr, diff;
   FILE* f;

   xcorr = (x + 3) >> 2 << 2;  // BMPs have to be a multiple of 4 pixels wide
   diff = xcorr - x;

   for (i = 0; i < 54; i++) hdr[i] = bmphdr[i];
   *((int*)(&hdr[18])) = xcorr;
   *((int*)(&hdr[22])) = y;
   *((int*)(&hdr[34])) = xcorr * y;
   *((int*)(&hdr[2])) = xcorr * y + 1078;
   for (i = 0; i < 256; i++) { 
      j = i * 4 + 54;
      hdr[j+0] = i;  // blue ColorTable
      hdr[j+1] = 0;  // green
      hdr[j+2] = 0;  // red
      hdr[j+3] = 0;  // dummy
   }

   f = fopen(name, "wb");  assert(f != NULL);
   c = fwrite(hdr, 1, 1078, f);  assert(c == 1078);
   if (diff == 0) {
      c = fwrite(bmp, 1, x * y, f);  assert(c == x * y);
   }
   else {
      *((int*)(&hdr[0])) = 0;  // need up to three zero bytes
      for (j = 0; j < y; j++) {
         c = fwrite(&bmp[j * x], 1, x, f);  assert(c == x);
         c = fwrite(hdr, 1, diff, f);  assert(c == diff);
      }
   }

   fclose(f);
}

int main(int argc, char *argv[]) {

   // check command line
   if (argc != 5) {
      fprintf(stderr, "usage: %s <frame_width> <num_frames> <num_blocks> <num_threads>\n", argv[0]);
      exit(-1);
   }
   int width = atoi(argv[1]);
   if (width < 100) {
      fprintf(stderr, "error: <frame_width> must be at least 100\n");
      exit(-1);
   }
   int frames = atoi(argv[2]);
   if (frames < 1) {
      fprintf(stderr, "error: <num_frames> must be at least 1\n");
      exit(-1);
   }
   int blocks = atoi(argv[3]);
   if (blocks < 1) {
      fprintf(stderr, "error: <num_blocks> must be at least 1\n");
      exit(-1);
   };
   int threads = atoi(argv[4]);
   if (threads < 1) {
      fprintf(stderr, "error: <num_threads> must be at least 1\n");
      exit(-1);
   };

   if (frames % blocks != 0) {
      fprintf(stderr, "Frames not divisible by number of blocks\n");
      exit(-1);
   }

   if (width % threads != 0) {
      fprintf(stderr, "Width size not divisible by number of threads\n");
      exit(-1);
   }

   printf("computing %d frames of %d by %d picture - %d blocks w/ %d threads.\n", frames, width, width, blocks, threads);

   // allocate picture array
   unsigned char* pic = NULL;
   hipMallocManaged(&pic, frames * width * width * sizeof(unsigned char));

   // start time
   timeval start, end;
   gettimeofday(&start, NULL);

   // Pixel Generator
   fPixelGenerator<<<blocks, threads>>>(width, frames, pic);
 
   // Wait for GPU to finish before accessing on host
   hipDeviceSynchronize();

   // end time
   gettimeofday(&end, NULL);
   double runtime = end.tv_sec + end.tv_usec / 1000000.0 - start.tv_sec - start.tv_usec / 1000000.0;
   printf("compute time: %.4f s\n", runtime);

   // verify result by writing frames to BMP files
   if ((width <= 256) && (frames <= 100)) {
      for (int frame = 0; frame < frames; frame++) {
         char name[32];
         sprintf(name, "wave%d.bmp", frame + 1000);
         writeBMP(width, width, &pic[frame * width * width], name);
      }
   }

   // Free memory
   hipFree(pic);

   return 0;
}