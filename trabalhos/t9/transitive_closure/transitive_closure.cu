
#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdlib>
#include <stdio.h>
#include <math.h>
#include <assert.h>
#include <sys/time.h>

__global__ void warshall(int nNodes, short int* graph) {

   int blockID = blockIdx.x;
   int tBlocks = gridDim.x;

   int threadID = threadIdx.x;
   int tThreads = blockDim.x;

   for(int k = blockID; k < nNodes; k += tBlocks) {
      for(int i = threadID; i < nNodes; i += tThreads) {
         for (int j = 0; j < nNodes; j++){

            if(graph[i * nNodes + k] + graph[k * nNodes + j] < graph[i * nNodes + j])
               graph[i * nNodes + j] = 1;

         }
      }
   }
}

long wtime() {
   struct timeval t;
   gettimeofday(&t, NULL);
   return t.tv_sec * 1000000 + t.tv_usec;
}

int read(short int** graph) {

   short int* aux_graph = NULL;
   int nNodes;

   char line[50];
   char* token;
   int size = 50;

   int l;
   int c;

   fgets(line, size, stdin);

   while(!feof(stdin)) {
      token = strtok(line," "); // split using space as divider
      if(*token == 'p') {

         token = strtok(NULL," "); // sp

         token  = strtok(NULL," "); // no. of vertices
         nNodes = atoi(token);

         token = strtok(NULL," "); // no. of directed edges

         // allocate picture array
         hipMallocManaged(&aux_graph, nNodes * nNodes * sizeof(short int));
         if (aux_graph == NULL) {
            printf("Error in graph allocation: NULL!\n");
            exit(EXIT_FAILURE);
         }

         for(int i = 0; i < nNodes;i++){
            for(int j = 0; j < nNodes;j++){
               aux_graph[i*nNodes+j] = 0;
            }
         }
      }
      else if(*token == 'a') {
         token = strtok(NULL," ");
         l = atoi(token)-1;

         token = strtok(NULL," ");
         c = atoi(token)-1;

         token = strtok(NULL," ");
         aux_graph[l*nNodes+c] = 1;
      }

      fgets(line, size, stdin);
   }

   *(graph) = aux_graph;

   return nNodes;
}

void write(int nNodes, short int* graph) {
   int i, j;

   for(i = 0; i < nNodes; i++) {
      for(j = 0; j < nNodes; j++) {
         printf("%d ", graph[i * nNodes + j]);
      }
      printf("\n");
   }
}


int main(int argc, char *argv[]) {

   int debug = 0;
   if(argc > 1)
      debug = atoi(argv[1]);

   short int* graph  = NULL;
   int        nNodes = read(&graph);

   // start time
   long start_time = wtime();

   // Warshall()
   warshall<<<2, 1024>>>(nNodes, graph);

   // Wait for GPU to finish before accessing on host
   hipDeviceSynchronize();

   // end time
   long end_time  = wtime();

   printf("warshall(): %ld usec\n", (long)(end_time - start_time));

   if(debug)
      write(nNodes, graph);

   hipFree(graph);

   return 0;
}